#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Cornerstone octree GPU testing
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 *
 */

#include "gtest/gtest.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "coord_samples/random.hpp"
#include "cstone/tree/csarray.hpp"
#include "cstone/tree/csarray_gpu.h"
#include "cstone/tree/cs_util.hpp"
#include "cstone/tree/update_gpu.cuh"

using namespace cstone;

TEST(CsArrayGpu, computeNodeCountsGpu)
{
    using KeyType = unsigned;

    // regular level-3 cornerstone tree with 512 leaves
    thrust::host_vector<KeyType> h_cstree = makeUniformNLevelTree<KeyType>(8 * 8 * 8, 1);
    // subdivide the first level-3 node
    for (int octant = 1; octant < 8; ++octant)
    {
        h_cstree.push_back(octant * nodeRange<KeyType>(4));
    }

    std::sort(begin(h_cstree), end(h_cstree));

    // create + upload tree to the device
    thrust::device_vector<KeyType> d_cstree = h_cstree;

    thrust::host_vector<KeyType> h_particleKeys;
    for (int nodeIdx = 1; nodeIdx < nNodes(h_cstree) - 1; ++nodeIdx)
    {
        // put 2 particles in each tree node, except the first and last node
        h_particleKeys.push_back(h_cstree[nodeIdx]);
        h_particleKeys.push_back(h_cstree[nodeIdx] + 1);
    }

    // upload particle codes to device
    thrust::device_vector<KeyType> d_particleKeys = h_particleKeys;

    thrust::device_vector<unsigned> d_counts(nNodes(d_cstree), 1);

    thrust::host_vector<unsigned> refCounts(nNodes(d_cstree), 2);
    // first and last nodes are empty
    refCounts[0]        = 0;
    *refCounts.rbegin() = 0;

    computeNodeCountsGpu(rawPtr(d_cstree), rawPtr(d_counts), nNodes(d_cstree), rawPtr(d_particleKeys),
                         rawPtr(d_particleKeys) + d_particleKeys.size(), std::numeric_limits<unsigned>::max(), false);
    thrust::host_vector<unsigned> h_counts = d_counts;
    EXPECT_EQ(h_counts, refCounts);

    // check again, using previous counts as guesses
    computeNodeCountsGpu(rawPtr(d_cstree), rawPtr(d_counts), nNodes(d_cstree), rawPtr(d_particleKeys),
                         rawPtr(d_particleKeys) + d_particleKeys.size(), std::numeric_limits<unsigned>::max(), true);
    h_counts = d_counts;
    EXPECT_EQ(h_counts, refCounts);
}

TEST(CsArrayGpu, rebalanceDecision)
{
    using KeyType       = unsigned;
    unsigned bucketSize = 8;

    thrust::device_vector<KeyType> tree = OctreeMaker<KeyType>{}.divide().divide(7).makeTree();
    thrust::device_vector<unsigned> counts(nNodes(tree), 1);
    counts[1] = 9;
    thrust::fill_n(counts.begin() + 8, 7, 0);

    thrust::device_vector<TreeNodeIndex> nodeOps(tree.size());
    computeNodeOpsGpu(rawPtr(tree), nNodes(tree), rawPtr(counts), bucketSize, rawPtr(nodeOps));

    // regular level-3 cornerstone tree with 512 leaves
    thrust::host_vector<TreeNodeIndex> h_nodeOps = nodeOps;

    thrust::host_vector<TreeNodeIndex> refNodeOps =
        std::vector<TreeNodeIndex>{0, 1, 9, 10, 11, 12, 13, 14, 15, 15, 15, 15, 15, 15, 15, 15};

    EXPECT_EQ(refNodeOps, h_nodeOps);
}

TEST(CsArrayGpu, rebalanceTree)
{
    using KeyType                       = unsigned;
    thrust::device_vector<KeyType> tree = OctreeMaker<KeyType>{}.divide().divide(7).makeTree();

    // node {1} to be split, nodes {7,i} are to be fused
    thrust::device_vector<TreeNodeIndex> nodeOps =
        std::vector<TreeNodeIndex>{0, 1, 9, 10, 11, 12, 13, 14, 15, 15, 15, 15, 15, 15, 15, 15};
    thrust::device_vector<KeyType> newTree(*nodeOps.rbegin() + 1);

    bool converged = rebalanceTreeGpu(rawPtr(tree), nNodes(tree), nNodes(newTree), rawPtr(nodeOps), rawPtr(newTree));

    // download tree from host
    thrust::host_vector<KeyType> h_tree    = newTree;
    thrust::host_vector<KeyType> reference = OctreeMaker<KeyType>{}.divide().divide(1).makeTree();
    EXPECT_EQ(h_tree, reference);
    EXPECT_FALSE(converged);
}

/*! @brief fixture for octree tests based on random particle distributions
 *
 * @tparam KeyType         32- or 64-bit unsigned integer
 *
 * These tests are already integration tests strictly speaking. They can be seen
 * as the second line of defense in case the unit tests above (with minimal and explict reference data)
 * fail to catch an error.
 */
template<class KeyType>
class OctreeFixtureGpu
{
public:
    OctreeFixtureGpu(unsigned numParticles, unsigned bucketSize)
    {
        d_codes = makeRandomGaussianKeys<KeyType>(numParticles);

        d_tree   = std::vector<KeyType>{0, nodeRange<KeyType>(0)};
        d_counts = std::vector<unsigned>{numParticles};

        thrust::device_vector<KeyType> tmpTree;
        thrust::device_vector<TreeNodeIndex> workArray;

        while (!updateOctreeGpu(thrust::raw_pointer_cast(d_codes.data()),
                                thrust::raw_pointer_cast(d_codes.data() + d_codes.size()), bucketSize, d_tree, d_counts,
                                tmpTree, workArray))
            ;
    }

    thrust::device_vector<KeyType> d_tree;
    thrust::device_vector<KeyType> d_codes;
    thrust::device_vector<unsigned> d_counts;
};

//! @brief build tree from random particles and compare against CPU
TEST(CsArrayGpu, computeOctreeRandom)
{
    using Integer = unsigned;

    int nParticles = 100000;
    int bucketSize = 64;

    // compute octree starting from default uniform octree
    auto particleKeys         = makeRandomGaussianKeys<Integer>(nParticles);
    auto [treeCpu, countsCpu] = computeOctree(particleKeys.data(), particleKeys.data() + nParticles, bucketSize);

    OctreeFixtureGpu<Integer> fixt(nParticles, bucketSize);

    // download tree from device
    thrust::host_vector<Integer> h_tree     = fixt.d_tree;
    thrust::host_vector<Integer> refTreeCpu = treeCpu;

    thrust::host_vector<Integer> h_counts     = fixt.d_counts;
    thrust::host_vector<Integer> refCountsCpu = countsCpu;

    EXPECT_EQ(h_tree, refTreeCpu);
    EXPECT_EQ(h_counts, refCountsCpu);
}

/*! @brief simulation of distributed tree
 *
 * In distributed octrees, the executing rank only has a part of the particle SFC codes, such that
 * many nodes in the tree are empty. Here this is simulated by removing a large connected part of the particle codes
 * and recomputing the node counts based on this subset of particle codes. The non-zero node counts should stay the
 * same.
 */
TEST(CsArrayGpu, distributedMockUp)
{
    using CodeType = unsigned;

    int nParticles = 100000;
    int bucketSize = 64;

    OctreeFixtureGpu<CodeType> fixt(nParticles, bucketSize);

    thrust::device_vector<CodeType> d_counts_orig = fixt.d_counts;

    // omit first and last tenth of nodes
    TreeNodeIndex Nodes     = nNodes(fixt.d_tree);
    TreeNodeIndex firstNode = Nodes / 10;
    TreeNodeIndex lastNode  = Nodes - Nodes / 10;

    // determine the part of the tree that will be empty
    thrust::host_vector<CodeType> h_codes = fixt.d_codes;
    unsigned firstParticleIdx =
        stl::lower_bound(h_codes.begin(), h_codes.end(), fixt.d_tree[firstNode]) - h_codes.begin();
    unsigned lastParticleIdx =
        stl::lower_bound(h_codes.begin(), h_codes.end(), fixt.d_tree[lastNode]) - h_codes.begin();
    std::cout << firstNode << " " << lastNode << std::endl;
    std::cout << firstParticleIdx << " " << lastParticleIdx << std::endl;

    bool useCountsAsGuess = true;
    computeNodeCountsGpu(thrust::raw_pointer_cast(fixt.d_tree.data()), thrust::raw_pointer_cast(fixt.d_counts.data()),
                         nNodes(fixt.d_tree), thrust::raw_pointer_cast(fixt.d_codes.data() + firstParticleIdx),
                         thrust::raw_pointer_cast(fixt.d_codes.data() + lastParticleIdx),
                         std::numeric_limits<unsigned>::max(), useCountsAsGuess);

    thrust::device_vector<CodeType> d_counts_ref = d_counts_orig;
    thrust::fill(d_counts_ref.begin(), d_counts_ref.begin() + firstNode, 0);
    thrust::fill(d_counts_ref.begin() + lastNode, d_counts_ref.end(), 0);

    EXPECT_EQ(fixt.d_counts, d_counts_ref);
}
