#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <chrono>
#include <numeric>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/cuda/gpu_config.cuh"

#include "nbody/dataset.hpp"
#include "ryoanji/interface/treebuilder.cuh"
#include "ryoanji/nbody/types.h"
#include "ryoanji/nbody/traversal.cuh"
#include "ryoanji/nbody/direct.cuh"
#include "ryoanji/nbody/upwardpass.cuh"

using namespace ryoanji;

int main(int argc, char** argv)
{
    constexpr int P     = 4;
    using T             = float;
    using MultipoleType = SphericalMultipole<T, P>;

    int power     = argc > 1 ? std::stoi(argv[1]) : 17;
    int directRef = argc > 2 ? std::stoi(argv[2]) : 1;

    std::size_t numBodies = (1 << power) - 1;
    T           theta     = 0.6;
    T           boxSize   = 3;
    T           G         = 1.0;

    const int ncrit = 64;

    fprintf(stdout, "--- BH Parameters ---------------\n");
    fprintf(stdout, "numBodies            : %lu\n", numBodies);
    fprintf(stdout, "P                    : %d\n", P);
    fprintf(stdout, "theta                : %f\n", theta);
    fprintf(stdout, "ncrit                : %d\n", ncrit);

    thrust::host_vector<T> x(numBodies), y(numBodies), z(numBodies), m(numBodies), h(numBodies);
    makeCubeBodies(x.data(), y.data(), z.data(), m.data(), h.data(), numBodies, boxSize);

    // upload bodies to device
    thrust::device_vector<T> d_x = x, d_y = y, d_z = z, d_m = m, d_h = h;

    cstone::Box<T> box(-boxSize, boxSize);

    TreeBuilder<uint64_t> treeBuilder;
    int                   numSources = treeBuilder.update(rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), numBodies, box);

    std::vector<int2> levelRange(treeBuilder.maxTreeLevel() + 1);
    int               highestLevel = treeBuilder.extract(levelRange.data());

    thrust::device_vector<Vec4<T>>       sourceCenter(numSources);
    thrust::device_vector<MultipoleType> Multipole(numSources);

    upsweep(numSources, treeBuilder.numLeafNodes(), highestLevel, theta, levelRange.data(), rawPtr(d_x), rawPtr(d_y),
            rawPtr(d_z), rawPtr(d_m), rawPtr(d_h), treeBuilder.layout(), treeBuilder.childOffsets(),
            treeBuilder.leafToInternal(), rawPtr(sourceCenter), rawPtr(Multipole));

    thrust::device_vector<T> d_p(numBodies, 0), d_ax(numBodies, 0), d_ay(numBodies, 0), d_az(numBodies, 0);

    fprintf(stdout, "--- BH Profiling ----------------\n");

    auto t0 = std::chrono::high_resolution_clock::now();

    auto interactions = computeAcceleration(0, numBodies, rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_m),
                                            rawPtr(d_h), G, rawPtr(d_p), rawPtr(d_ax), rawPtr(d_ay), rawPtr(d_az),
                                            treeBuilder.childOffsets(), treeBuilder.internalToLeaf(),
                                            treeBuilder.layout(), rawPtr(sourceCenter), rawPtr(Multipole));

    auto   t1    = std::chrono::high_resolution_clock::now();
    double dt    = std::chrono::duration<double>(t1 - t0).count();
    double flops = (interactions[0] * 20 + interactions[2] * 2 * pow(P, 3)) * numBodies / dt / 1e12;

    fprintf(stdout, "--- Total runtime ----------------\n");
    fprintf(stdout, "Total BH            : %.7f s (%.7f TFlops)\n", dt, flops);

    if (!directRef) { return 0; }

    thrust::device_vector<T> refP(numBodies), refAx(numBodies), refAy(numBodies), refAz(numBodies);

    t0 = std::chrono::high_resolution_clock::now();
    directSum(0, numBodies, numBodies, rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_m), rawPtr(d_h), rawPtr(refP),
              rawPtr(refAx), rawPtr(refAy), rawPtr(refAz));

    t1 = std::chrono::high_resolution_clock::now();
    dt = std::chrono::duration<double>(t1 - t0).count();

    flops = 24. * numBodies * numBodies / dt / 1e12;
    fprintf(stdout, "Total Direct         : %.7f s (%.7f TFlops)\n", dt, flops);

    thrust::host_vector<T> h_p  = d_p;
    thrust::host_vector<T> h_ax = d_ax;
    thrust::host_vector<T> h_ay = d_ay;
    thrust::host_vector<T> h_az = d_az;

    double                 referencePotential = 0.5 * G * thrust::reduce(refP.begin(), refP.end(), 0.0);
    thrust::host_vector<T> h_refAx            = refAx;
    thrust::host_vector<T> h_refAy            = refAy;
    thrust::host_vector<T> h_refAz            = refAz;

    std::vector<double> delta(numBodies);

    double potentialSum = 0;
    for (int i = 0; i < numBodies; i++)
    {
        potentialSum += h_p[i];
        Vec3<T> ref   = {h_refAx[i], h_refAy[i], h_refAz[i]};
        Vec3<T> probe = {h_ax[i], h_ay[i], h_az[i]};
        delta[i]      = std::sqrt(norm2(ref - probe) / norm2(ref));
    }

    std::sort(begin(delta), end(delta));

    fprintf(stdout, "--- BH vs. direct ---------------\n");

    std::cout << "potentials, body-sum: " << 0.5 * G * potentialSum << " atomic sum: " << 0.5 * G * interactions[4]
              << " reference: " << referencePotential << std::endl;
    std::cout << "min Error: " << delta[0] << std::endl;
    std::cout << "50th percentile: " << delta[numBodies / 2] << std::endl;
    std::cout << "10th percentile: " << delta[numBodies * 0.9] << std::endl;
    std::cout << "1st percentile: " << delta[numBodies * 0.99] << std::endl;
    std::cout << "max Error: " << delta[numBodies - 1] << std::endl;

    fprintf(stdout, "--- Tree stats -------------------\n");
    fprintf(stdout, "Bodies               : %lu\n", numBodies);
    fprintf(stdout, "Cells                : %d\n", numSources);
    fprintf(stdout, "Tree depth           : %d\n", highestLevel);
    fprintf(stdout, "--- Traversal stats --------------\n");
    fprintf(stdout, "P2P mean list length : %d (max %d)\n", int(interactions[0]), int(interactions[1]));
    fprintf(stdout, "M2P mean list length : %d (max %d)\n", int(interactions[2]), int(interactions[3]));

    return 0;
}
