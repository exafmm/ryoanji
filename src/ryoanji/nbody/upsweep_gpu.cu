#include "hip/hip_runtime.h"
/*
 * Ryoanji N-body solver
 *
 * Copyright (c) 2024 CSCS, ETH Zurich
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: MIT License
 */

/*! @file
 * @brief  Upsweep for multipole and source center computation
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_runtime.hpp"
#include "cstone/primitives/math.hpp"

#include "ryoanji/nbody/cartesian_qpole.hpp"
#include "ryoanji/nbody/kernel.hpp"

#include "upsweep_gpu.h"

namespace ryoanji
{

struct UpsweepConfig
{
    static constexpr int numThreads = 256;
};

template<class Tc, class Tm, class Tf, class MType>
__global__ void computeLeafMultipolesKernel(const Tc* x, const Tc* y, const Tc* z, const Tm* m,
                                            const TreeNodeIndex* leafToInternal, TreeNodeIndex numLeaves,
                                            const LocalIndex* layout, const Vec4<Tf>* centers, MType* multipoles)
{
    unsigned leafIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (leafIdx < numLeaves)
    {
        TreeNodeIndex i = leafToInternal[leafIdx];
        P2M(x, y, z, m, layout[leafIdx], layout[leafIdx + 1], centers[i], multipoles[i]);
    }
}

template<class Tc, class Tm, class Tf, class MType>
void computeLeafMultipoles(const Tc* x, const Tc* y, const Tc* z, const Tm* m, const TreeNodeIndex* leafToInternal,
                           TreeNodeIndex numLeaves, const LocalIndex* layout, const Vec4<Tf>* centers,
                           MType* multipoles)
{
    constexpr int numThreads = UpsweepConfig::numThreads;
    if (numLeaves)
    {
        computeLeafMultipolesKernel<<<cstone::iceil(numLeaves, numThreads), numThreads>>>(
            x, y, z, m, leafToInternal, numLeaves, layout, centers, multipoles);
    }
}

#define COMPUTE_LEAF_MULTIPOLES(Tc, Tm, Tf, MType)                                                                     \
    template void computeLeafMultipoles(const Tc* x, const Tc* y, const Tc* z, const Tm* m,                            \
                                        const TreeNodeIndex* leafToInternal, TreeNodeIndex numLeaves,                  \
                                        const LocalIndex* layout, const Vec4<Tf>* centers, MType* multipoles)

template<class T, class MType>
__global__ void upsweepMultipolesKernel(TreeNodeIndex firstCell, TreeNodeIndex lastCell,
                                        const TreeNodeIndex* childOffsets, const Vec4<T>* centers, MType* multipoles)
{
    const int cellIdx = blockIdx.x * blockDim.x + threadIdx.x + firstCell;
    if (cellIdx >= lastCell) return;

    TreeNodeIndex firstChild = childOffsets[cellIdx];

    // firstChild is zero if the cell is a leaf
    if (firstChild) { M2M(firstChild, firstChild + 8, centers[cellIdx], centers, multipoles, multipoles[cellIdx]); }
}

template<class T, class MType>
void upsweepMultipoles(TreeNodeIndex firstCell, TreeNodeIndex lastCell, const TreeNodeIndex* childOffsets,
                       const Vec4<T>* centers, MType* multipoles)
{
    constexpr int numThreads = UpsweepConfig::numThreads;
    if (lastCell > firstCell)
    {
        upsweepMultipolesKernel<<<cstone::iceil(lastCell - firstCell, numThreads), numThreads>>>(
            firstCell, lastCell, childOffsets, centers, multipoles);
    }
}

#define UPSWEEP_MULTIPOLES(T, MType)                                                                                   \
    template void upsweepMultipoles(TreeNodeIndex firstCell, TreeNodeIndex lastCell,                                   \
                                    const TreeNodeIndex* childOffsets, const Vec4<T>* centers, MType* multipoles)

#define INSTANTIATE_MULTIPOLE(MType)                                                                                   \
    COMPUTE_LEAF_MULTIPOLES(double, double, double, MType<double>);                                                    \
    COMPUTE_LEAF_MULTIPOLES(double, float, double, MType<float>);                                                      \
    COMPUTE_LEAF_MULTIPOLES(float, float, float, MType<float>);                                                        \
    UPSWEEP_MULTIPOLES(double, MType<double>);                                                                         \
    UPSWEEP_MULTIPOLES(double, MType<float>);                                                                          \
    UPSWEEP_MULTIPOLES(float, MType<float>);

INSTANTIATE_MULTIPOLE(CartesianQuadrupole)
INSTANTIATE_MULTIPOLE(CartesianMDQpole)

} // namespace ryoanji
