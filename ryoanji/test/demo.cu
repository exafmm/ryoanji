#include "hip/hip_runtime.h"
/*
 * Ryoanji N-body solver
 *
 * Copyright (c) 2024 CSCS, ETH Zurich
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: MIT License
 */

/*! @file
 * @brief  Single-GPU demonstrator app for the Ryoanji N-body library
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 * @author Rio Yokota <rioyokota@gsic.titech.ac.jp>
 */

#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/cuda/gpu_config.cuh"
#include "cstone/cuda/thrust_util.cuh"
#include "cstone/focus/source_center_gpu.h"
#include "cstone/traversal/groups_gpu.h"
#include "cstone/util/array.hpp"

#include "nbody/dataset.hpp"
#include "ryoanji/interface/treebuilder.cuh"
#include "ryoanji/nbody/types.h"
#include "ryoanji/nbody/cartesian_qpole.hpp"
#include "ryoanji/nbody/traversal_gpu.h"
#include "ryoanji/nbody/direct.cuh"
#include "ryoanji/nbody/upsweep_gpu.h"

using namespace ryoanji;

template<class Tc, class Th, class Tm, class Ta, class Tf, class MType>
util::array<Tc, 5> computeAcceleration(size_t firstBody, size_t lastBody, const Tc* x, const Tc* y, const Tc* z,
                                       const Tm* m, const Th* h, Tc G, int numShells, const cstone::Box<Tc>& box, Ta* p,
                                       Ta* ax, Tc* ay, Tc* az, const TreeNodeIndex* childOffsets,
                                       const TreeNodeIndex* internalToLeaf, const LocalIndex* layout,
                                       const Vec4<Tf>* sourceCenter, const MType* Multipole);

template<class KeyType, class T, class MType>
void upsweep(int numSources, int numLeaves, int numLevels, float theta, const TreeNodeIndex* levelRange, const T* x,
             const T* y, const T* z, const T* m, const cstone::Box<T>& box, const LocalIndex* layout,
             const KeyType* prefixes, const TreeNodeIndex* childOffsets, const TreeNodeIndex* leafToInternal,
             Vec4<T>* centers, MType* Multipole);

int main(int argc, char** argv)
{
    using T             = float;
    using MultipoleType = CartesianQuadrupole<T>;

    int power     = argc > 1 ? std::stoi(argv[1]) : 17;
    int directRef = argc > 2 ? std::stoi(argv[2]) : 1;
    int numShells = argc > 3 ? std::stoi(argv[3]) : 0;

    std::size_t numBodies = (1 << power) - 1;
    T           theta     = 0.6;
    T           boxSize   = 3;
    T           G         = 1.0;

    const int ncrit = 64;

    fprintf(stdout, "--- BH Parameters ---------------\n");
    fprintf(stdout, "numBodies            : %lu\n", numBodies);
    fprintf(stdout, "theta                : %f\n", theta);
    fprintf(stdout, "ncrit                : %d\n", ncrit);

    thrust::host_vector<T> x(numBodies), y(numBodies), z(numBodies), m(numBodies), h(numBodies);
    makeCubeBodies(x.data(), y.data(), z.data(), m.data(), h.data(), numBodies, boxSize);

    // upload bodies to device
    thrust::device_vector<T> d_x = x, d_y = y, d_z = z, d_m = m, d_h = h;

    cstone::Box<T> box(-boxSize, boxSize);

    TreeBuilder<uint64_t> treeBuilder(ncrit);
    int                   numSources = treeBuilder.update(rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), numBodies, box);

    const TreeNodeIndex* levelRange   = treeBuilder.levelRange();
    int                  highestLevel = treeBuilder.maxTreeLevel();

    thrust::device_vector<Vec4<T>>       sourceCenter(numSources);
    thrust::device_vector<MultipoleType> Multipole(numSources);

    upsweep(numSources, treeBuilder.numLeafNodes(), highestLevel, theta, levelRange, rawPtr(d_x), rawPtr(d_y),
            rawPtr(d_z), rawPtr(d_m), box, treeBuilder.layout(), treeBuilder.nodeKeys(), treeBuilder.childOffsets(),
            treeBuilder.leafToInternal(), rawPtr(sourceCenter), rawPtr(Multipole));

    thrust::device_vector<T> d_p(numBodies, 0), d_ax(numBodies, 0), d_ay(numBodies, 0), d_az(numBodies, 0);

    fprintf(stdout, "--- BH Profiling ----------------\n");

    auto t0 = std::chrono::high_resolution_clock::now();

    auto interactions = computeAcceleration(0, numBodies, rawPtr(d_x), rawPtr(d_y), rawPtr(d_z), rawPtr(d_m),
                                            rawPtr(d_h), G, numShells, box, rawPtr(d_p), rawPtr(d_ax), rawPtr(d_ay),
                                            rawPtr(d_az), treeBuilder.childOffsets(), treeBuilder.internalToLeaf(),
                                            treeBuilder.layout(), rawPtr(sourceCenter), rawPtr(Multipole));

    auto   t1    = std::chrono::high_resolution_clock::now();
    double dt    = std::chrono::duration<double>(t1 - t0).count();
    double flops = (interactions[0] * 23 + interactions[2] * 65) * numBodies / dt / 1e12;

    fprintf(stdout, "--- Total runtime ----------------\n");
    fprintf(stdout, "Total BH            : %.7f s (%.7f TFlops)\n", dt, flops);

    if (!directRef) { return 0; }

    thrust::device_vector<T> refP(numBodies), refAx(numBodies), refAy(numBodies), refAz(numBodies);

    t0 = std::chrono::high_resolution_clock::now();
    directSum(0, numBodies, numBodies, Vec3<T>{box.lx(), box.ly(), box.lz()}, numShells, rawPtr(d_x), rawPtr(d_y),
              rawPtr(d_z), rawPtr(d_m), rawPtr(d_h), rawPtr(refP), rawPtr(refAx), rawPtr(refAy), rawPtr(refAz));

    t1 = std::chrono::high_resolution_clock::now();
    dt = std::chrono::duration<double>(t1 - t0).count();

    flops = std::pow((2 * numShells + 1), 3) * 23. * numBodies * numBodies / dt / 1e12;
    fprintf(stdout, "Total Direct         : %.7f s (%.7f TFlops)\n", dt, flops);

    thrust::host_vector<T> h_p  = d_p;
    thrust::host_vector<T> h_ax = d_ax;
    thrust::host_vector<T> h_ay = d_ay;
    thrust::host_vector<T> h_az = d_az;

    double                 referencePotential = 0.5 * G * thrust::reduce(refP.begin(), refP.end(), 0.0);
    thrust::host_vector<T> h_refAx            = refAx;
    thrust::host_vector<T> h_refAy            = refAy;
    thrust::host_vector<T> h_refAz            = refAz;

    std::vector<double> delta(numBodies);

    double potentialSum = 0;
    for (int i = 0; i < numBodies; i++)
    {
        potentialSum += h_p[i];
        Vec3<T> ref   = {h_refAx[i], h_refAy[i], h_refAz[i]};
        Vec3<T> probe = {h_ax[i], h_ay[i], h_az[i]};
        delta[i]      = std::sqrt(norm2(ref - probe) / norm2(ref));
    }

    std::sort(begin(delta), end(delta));

    fprintf(stdout, "--- BH vs. direct ---------------\n");

    std::cout << "potentials, body-sum: " << 0.5 * G * potentialSum << " atomic sum: " << interactions[4]
              << " reference: " << referencePotential << std::endl;
    std::cout << "min Error: " << delta[0] << std::endl;
    std::cout << "50th percentile: " << delta[numBodies / 2] << std::endl;
    std::cout << "10th percentile: " << delta[numBodies * 0.9] << std::endl;
    std::cout << "1st percentile: " << delta[numBodies * 0.99] << std::endl;
    std::cout << "max Error: " << delta[numBodies - 1] << std::endl;

    fprintf(stdout, "--- Tree stats -------------------\n");
    fprintf(stdout, "Bodies               : %lu\n", numBodies);
    fprintf(stdout, "Cells                : %d\n", numSources);
    fprintf(stdout, "Tree depth           : %d\n", highestLevel);
    fprintf(stdout, "--- Traversal stats --------------\n");
    fprintf(stdout, "P2P mean list length : %d (max %d)\n", int(interactions[0]), int(interactions[1]));
    fprintf(stdout, "M2P mean list length : %d (max %d)\n", int(interactions[2]), int(interactions[3]));

    return 0;
}

template<class Tc, class Th, class Tm, class Ta, class Tf, class MType>
util::array<Tc, 5> computeAcceleration(size_t firstBody, size_t lastBody, const Tc* x, const Tc* y, const Tc* z,
                                       const Tm* m, const Th* h, Tc G, int numShells, const cstone::Box<Tc>& box, Ta* p,
                                       Ta* ax, Tc* ay, Tc* az, const TreeNodeIndex* childOffsets,
                                       const TreeNodeIndex* internalToLeaf, const LocalIndex* layout,
                                       const Vec4<Tf>* sourceCenter, const MType* Multipole)
{
    auto                              numBodies = lastBody - firstBody;
    cstone::GroupData<cstone::GpuTag> groups;
    cstone::computeFixedGroups(firstBody, lastBody, bhMaxTargetSize(), groups);
    thrust::device_vector<int> globalPool(stackSize(groups.numGroups));

    double totalPotential = traverse(groups.view(), 1, x, y, z, m, h, x, y, z, m, h, childOffsets, internalToLeaf,
                                     layout, sourceCenter, Multipole, G, numShells, {box.lx(), box.ly(), box.lz()}, p,
                                     ax, ay, az, thrust::raw_pointer_cast(globalPool.data()));
    kernelSuccess("traverse");

    auto stats = readBhStats();
    return {Tc(stats[0]) / numBodies, Tc(stats[1]), Tc(stats[2]) / numBodies, Tc(stats[3]), Tc(totalPotential)};
}

template<class KeyType, class T, class MType>
void upsweep(int numSources, int numLeaves, int numLevels, float theta, const TreeNodeIndex* levelRange, const T* x,
             const T* y, const T* z, const T* m, const cstone::Box<T>& box, const LocalIndex* layout,
             const KeyType* prefixes, const TreeNodeIndex* childOffsets, const TreeNodeIndex* leafToInternal,
             Vec4<T>* centers, MType* Multipole)
{
    auto t0 = std::chrono::high_resolution_clock::now();

    cstone::computeLeafSourceCenterGpu(x, y, z, m, leafToInternal, numLeaves, layout, centers);
    cstone::upsweepCentersGpu(cstone::maxTreeLevel<KeyType>{}, levelRange, childOffsets, centers);

    computeLeafMultipoles(x, y, z, m, leafToInternal, numLeaves, layout, centers, Multipole);
    for (int level = numLevels - 1; level >= 1; level--)
    {
        upsweepMultipoles(levelRange[level], levelRange[level + 1], childOffsets, centers, Multipole);
    }

    cstone::setMacGpu(prefixes, numSources, centers, 1.f / theta, box);

    auto   t1 = std::chrono::high_resolution_clock::now();
    double dt = std::chrono::duration<double>(t1 - t0).count();

    fprintf(stdout, "Upward pass          : %.7f s\n", dt);
}
