#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2024 CSCS, ETH Zurich, University of Basel, University of Zurich
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Upsweep for multipole and source center computation
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/cuda/cuda_runtime.hpp"
#include "cstone/primitives/math.hpp"

#include "ryoanji/nbody/cartesian_qpole.hpp"
#include "ryoanji/nbody/kernel.hpp"

#include "upsweep_gpu.h"

namespace ryoanji
{

struct UpsweepConfig
{
    static constexpr int numThreads = 256;
};

template<class Tc, class Tm, class Tf, class MType>
__global__ void computeLeafMultipolesKernel(const Tc* x, const Tc* y, const Tc* z, const Tm* m,
                                            const TreeNodeIndex* leafToInternal, TreeNodeIndex numLeaves,
                                            const LocalIndex* layout, const Vec4<Tf>* centers, MType* multipoles)
{
    unsigned leafIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (leafIdx < numLeaves)
    {
        TreeNodeIndex i = leafToInternal[leafIdx];
        P2M(x, y, z, m, layout[leafIdx], layout[leafIdx + 1], centers[i], multipoles[i]);
    }
}

template<class Tc, class Tm, class Tf, class MType>
void computeLeafMultipoles(const Tc* x, const Tc* y, const Tc* z, const Tm* m, const TreeNodeIndex* leafToInternal,
                           TreeNodeIndex numLeaves, const LocalIndex* layout, const Vec4<Tf>* centers,
                           MType* multipoles)
{
    constexpr int numThreads = UpsweepConfig::numThreads;
    if (numLeaves)
    {
        computeLeafMultipolesKernel<<<cstone::iceil(numLeaves, numThreads), numThreads>>>(
            x, y, z, m, leafToInternal, numLeaves, layout, centers, multipoles);
    }
}

#define COMPUTE_LEAF_MULTIPOLES(Tc, Tm, Tf, MType)                                                                     \
    template void computeLeafMultipoles(const Tc* x, const Tc* y, const Tc* z, const Tm* m,                            \
                                        const TreeNodeIndex* leafToInternal, TreeNodeIndex numLeaves,                  \
                                        const LocalIndex* layout, const Vec4<Tf>* centers, MType* multipoles)

template<class T, class MType>
__global__ void upsweepMultipolesKernel(TreeNodeIndex firstCell, TreeNodeIndex lastCell,
                                        const TreeNodeIndex* childOffsets, const Vec4<T>* centers, MType* multipoles)
{
    const int cellIdx = blockIdx.x * blockDim.x + threadIdx.x + firstCell;
    if (cellIdx >= lastCell) return;

    TreeNodeIndex firstChild = childOffsets[cellIdx];

    // firstChild is zero if the cell is a leaf
    if (firstChild) { M2M(firstChild, firstChild + 8, centers[cellIdx], centers, multipoles, multipoles[cellIdx]); }
}

template<class T, class MType>
void upsweepMultipoles(TreeNodeIndex firstCell, TreeNodeIndex lastCell, const TreeNodeIndex* childOffsets,
                       const Vec4<T>* centers, MType* multipoles)
{
    constexpr int numThreads = UpsweepConfig::numThreads;
    if (lastCell > firstCell)
    {
        upsweepMultipolesKernel<<<cstone::iceil(lastCell - firstCell, numThreads), numThreads>>>(
            firstCell, lastCell, childOffsets, centers, multipoles);
    }
}

#define UPSWEEP_MULTIPOLES(T, MType)                                                                                   \
    template void upsweepMultipoles(TreeNodeIndex firstCell, TreeNodeIndex lastCell,                                   \
                                    const TreeNodeIndex* childOffsets, const Vec4<T>* centers, MType* multipoles)

#define INSTANTIATE_MULTIPOLE(MType)                                                                                   \
    COMPUTE_LEAF_MULTIPOLES(double, double, double, MType<double>);                                                    \
    COMPUTE_LEAF_MULTIPOLES(double, float, double, MType<float>);                                                      \
    COMPUTE_LEAF_MULTIPOLES(float, float, float, MType<float>);                                                        \
    UPSWEEP_MULTIPOLES(double, MType<double>);                                                                         \
    UPSWEEP_MULTIPOLES(double, MType<float>);                                                                          \
    UPSWEEP_MULTIPOLES(float, MType<float>);

INSTANTIATE_MULTIPOLE(CartesianQuadrupole)
INSTANTIATE_MULTIPOLE(CartesianMDQpole)

} // namespace ryoanji
